﻿#include"core/SLICAP.h"








int main() {

	std::cout << "initializing engine" << std::endl;
	SLICAP engine("examples/example.png");

	std::cout << "running SLIC" << std::endl;
	engine.run_SLIC();

	std::cout << "displaying labels" << std::endl;
	engine.display_SLIC_result();

	std::cout << "running AP" << std::endl;
	engine.run_AP();

	std::cout << "displaying result" << std::endl;
	engine.display_AP_result();

	return 0;
}